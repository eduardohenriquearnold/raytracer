#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "ray.h"
#include "camera.h"
#include "hitable.h"
#include "material.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const ray& r, hitable** h, hiprandState *s) {
  ray cur_ray = r;
  vec3 cur_attenuation = vec3(1.,1.,1.);
  
  for (int iter=0; iter<50; iter++)
  {
    hit_record rec;
    if ((**h).hit(cur_ray, 0.001f, 1e20f, rec)) {
      ray scattered;
      vec3 attenuation;
      if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, s)) {
        cur_attenuation *= attenuation;
        cur_ray = scattered;
      }
      else 
        return vec3(0,0,0);
    }
    else{
      //background
      vec3 unit_direction = unit_vector(r.direction());
      float t = 0.5f*(unit_direction.y() + 1.0f);
      vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
      return cur_attenuation*c;
    }
  }

  //if exceeds max iteration (number of reflections)
  return vec3(0,0,0);
}

__global__ void render(vec3 *fb, const int nx, const int ny, const int ns, camera** cam, hitable** h, hiprandState *crs) {
  int i = blockIdx.x * blockDim.x +threadIdx.x;
  int j = blockIdx.y * blockDim.y +threadIdx.y;
  int pixelIdx = i + j*nx;
  if (i>=nx || j>=ny)
    return;
  hiprandState ls = crs[pixelIdx];

  vec3 col(0,0,0);
  for (int s=0; s<ns; s++)
  {
    float u = (i +random_float(&ls))/ float(nx);
    float v = (j +random_float(&ls)) / float(ny);
    ray r = (**cam).get_ray(u,v);
    col += color(r, h, &ls);
  }
  col /= ns;

  //gamma correction
  col[0] = clamp(sqrt(col[0]), 0, 0.999);
  col[1] = clamp(sqrt(col[1]), 0, 0.999);
  col[2] = clamp(sqrt(col[2]), 0, 0.999);
  col *= 256;
  fb[pixelIdx] = col;
}

__global__ void init_random_states(hiprandState *s, const int nx, const int ny) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= nx) || (j >= ny)) return;
  int pixel_index = i + j*nx;

  //Each thread gets different seed and same sequence number (more efficient than using same seed with different sequence number, according to NVIDIA docs)
  hiprand_init(1984+pixel_index, 0, 0, &s[pixel_index]);    
}

__global__ void init_cam(camera** c, int nx, int ny){
  if (!(threadIdx.x == 0 && blockIdx.x == 0))
    return;

  // camera *cam = new camera(vec3(13,2,3), vec3(0,0,0), vec3(0,1,0), 20, float(nx)/float(ny));
  *c = new camera(vec3(0,0,0), vec3(0,0,-1), vec3(0,1,0), 90, float(nx)/ny);
}

__global__ void init_world(hitable** w){
  if (!(threadIdx.x == 0 && blockIdx.x == 0))
    return;

  *w = new hitable_list(10);

  hitable_list* world = (hitable_list*) *w;
  world->add(new sphere(vec3(0,-100.5,-1), 100.f, new lambertian(vec3(0.3,0.3,0.3))));
  world->add(new sphere(vec3(0,0,-1), 0.5f, new lambertian(vec3(0.3,0.5,0.3)) ));
}

int main()
{
  std::ofstream f;
  f.open("output.ppm", std::ios::out);

  //define rendering limits/properties
  const int nx = 600;
  const int ny = 400;
  const int ns = 1000;

  //define thread numbers and block dimensions
  const int tx = 8;
  const int ty = 8;
  const dim3 blocks(nx/tx+1, ny/ty+1);
  const dim3 threads(tx,ty);

  //allocate device memory for image
  vec3 *fb;
  hipMallocManaged(&fb, sizeof(vec3)*nx*ny);

  //create random states
  hiprandState *crs;
  hipMalloc(&crs, nx*ny*sizeof(hiprandState));
  init_random_states<<<blocks,threads>>>(crs, nx, ny);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());

  //create camera
  camera **cam;
  hipMalloc(&cam, sizeof(camera*));
  init_cam<<<1,1>>>(cam, nx, ny);
  hipDeviceSynchronize();
  
  //create world (objects and materials) in device
  hitable** world;
  hipMalloc(&world, sizeof(hitable*));
  init_world<<<1,1>>>(world);
  hipDeviceSynchronize();

  //render world
  std::cout << "Started rendering..." << std::endl;
  render<<<blocks, threads>>>(fb, nx, ny, ns, cam, world, crs);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
  std::cout << "Finished rendering." << std::endl;

  //save image
  f << "P3\n" << nx << " " << ny << "\n255\n";
  for (int j=ny-1; j>=0; j--)
    for (int i=0; i<nx; i++)
    {
      vec3 col = fb[i+j*nx];
      f << int(col.e[0]) << " " << int(col.e[1]) << " " << int(col.e[2]) << "\n";
    }
  f.close();
  std::cout << "Saved image." << std::endl;

  //free memory
  hipFree(fb);
  hipFree(crs);
  hipFree(cam);
  hipFree(world);
}
